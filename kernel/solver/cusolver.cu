// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>

#include <kernel/util/exception.hpp>

#include <hipsolver.h>
#include "hipsparse.h"

using namespace FEAT;

namespace FEAT
{
  namespace Solver
  {
    namespace Intern
    {
      int cuda_lu(int n, int nnzA, const double * csrValA, const int * csrRowPtrA, const int * csrColIndA,
          const double * b, double * x)
      {
        hipsolverSpHandle_t handle;
        hipsolverSpCreate(&handle);

        hipsparseMatDescr_t descr;
        hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        int singularity;
        hipsolverStatus_t status = cusolverSpDcsrlsvluHost(handle, n, nnzA, descr, csrValA, csrRowPtrA, csrColIndA, b, 0.0, 1, x, &singularity);
        if (status != HIPSOLVER_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusolverSpDcsrlsvluHost failed with status code: " + stringify(status));

        hipsparseDestroyMatDescr(descr);
        hipsolverSpDestroy(handle);

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return (status != HIPSOLVER_STATUS_SUCCESS);
      }


      int cuda_qr(int m, int nnz, const double * csrValA, const int * csrRowPtrA, const int * csrColIndA,
          const double * b, double * x)
      {
        hipsolverSpHandle_t handle;
        hipsolverSpCreate(&handle);

        hipsparseMatDescr_t descr;
        hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        int singularity;
        hipsolverStatus_t status = hipsolverSpDcsrlsvqr(handle, m, nnz, descr, csrValA, csrRowPtrA, csrColIndA, b, 0.0, 1, x, &singularity);
        if (status != HIPSOLVER_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusolverSPDcsrlvsqr failed with status code: " + stringify(status));

        hipsparseDestroyMatDescr(descr);
        hipsolverSpDestroy(handle);

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return (status != HIPSOLVER_STATUS_SUCCESS);
      }
    }
  }
}
