#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2023 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>

#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

#include "hipsparse.h"

// http://docs.nvidia.com/cuda/cusparse/#cusparse-lt-t-gt-csrilu02_solve

using namespace FEAT;

namespace FEAT
{
  namespace Solver
  {
    /// \cond internal
    namespace Intern
    {
      // CSR
      struct CudaIluSolveInfo
      {
        hipsparseMatDescr_t descr_M;
#if CUSPARSE_VER_MAJOR < 12
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
#else
        hipsparseSpMatDescr_t descr_L;
        hipsparseSpMatDescr_t descr_U;
        //in this case, we also need handler for input and output vectors
        //hipsparseDnVecDescr_t descr_X;  //should not be needed, since buffersize and anaylsis accept NULL as vectors...
        //hipsparseDnVecDescr_t descr_Y;

#endif
        csrilu02Info_t info_M;
#if CUSPARSE_VER_MAJOR < 12
        csrsv2Info_t  info_L;
        csrsv2Info_t  info_U;
#else
        hipsparseSpSVDescr_t  info_L;
        hipsparseSpSVDescr_t  info_U;
#endif

        int pBufferSize_M;
#if CUSPARSE_VER_MAJOR < 12
        int pBufferSize_L;
        int pBufferSize_U;
#else
        size_t pBufferSize_L;
        size_t pBufferSize_U;
#endif
        int pBufferSize;
        void *pBuffer;
        int structural_zero;
        int numerical_zero;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL; //why?
        double * z;
        int m;
        int nnz;
      };

      void * cuda_ilu_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd)
      {
        CudaIluSolveInfo * info = new CudaIluSolveInfo;
        info->m = m;
        info->nnz = nnz;

        info->z = (double*)Util::cuda_malloc(m * sizeof(double));


        hipsparseStatus_t status;

        hipsparseCreateMatDescr(&(info->descr_M));
        hipsparseSetMatIndexBase(info->descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(info->descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

#if CUSPARSE_VER_MAJOR < 12
        hipsparseCreateMatDescr(&(info->descr_L));
        hipsparseSetMatIndexBase(info->descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(info->descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(info->descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(info->descr_L, HIPSPARSE_DIAG_TYPE_UNIT);
#else
        //assertion if int is 32 bits
        static_assert(sizeof(int) == 4u, "ERROR: Size of int is not 32 bits");
        hipsparseCreateCsr(&(info->descr_L), info->m, info->m, info->nnz,
                                      csrRowPtr, csrColInd, csrVal,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,   //use typedef somewhere? Since this goes wrong, if int is something other than 32 bits...
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);     //Also variable size in theroy...
        //set attributes
        {
          hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_LOWER;
          hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
          hipsparseSpMatSetAttribute(info->descr_L, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(hipsparseFillMode_t)); //set relevant data, rest should be set by default due to new CSR implementation...
          hipsparseSpMatSetAttribute(info->descr_L, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(hipsparseSpMatAttribute_t));
        }
#endif

#if CUSPARSE_VER_MAJOR < 12
        hipsparseCreateMatDescr(&(info->descr_U));
        hipsparseSetMatIndexBase(info->descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(info->descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(info->descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(info->descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
#else
        hipsparseCreateCsr(&(info->descr_U), info->m, info->m, info->nnz,
                                      csrRowPtr, csrColInd, csrVal,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
        {
          hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_UPPER;
          hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
          hipsparseSpMatSetAttribute(info->descr_U, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(hipsparseFillMode_t)); //set relevant data, rest should be set by default due to new CSR implementation...
          hipsparseSpMatSetAttribute(info->descr_U, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(hipsparseSpMatAttribute_t));
        }
#endif

        hipsparseCreateCsrilu02Info(&(info->info_M));
#if CUSPARSE_VER_MAJOR < 12
        hipsparseCreateCsrsv2Info(&(info->info_L));
        hipsparseCreateCsrsv2Info(&(info->info_U));
#else
        // create information handler for cuSparseSolver
        hipsparseSpSV_createDescr(&(info->info_L));
        hipsparseSpSV_createDescr(&(info->info_U));
#endif

// #if CUSPARSE_VER_MAJOR >= 12
//         //for now, we need to create pseudo vector arrays... we will set these later to the real vector by transfering the data pointer
//         hipsparseCreateDnVec(&(info->descr_X), info->m, nullptr, HIP_R_64F);
//         hipsparseCreateDnVec(&(info->descr_Y), info->m, nullptr, HIP_R_64F);
// #endif

        status = hipsparseDcsrilu02_bufferSize(Util::Intern::cusparse_handle, m, nnz,
                info->descr_M, csrVal, csrRowPtr, csrColInd, info->info_M, &(info->pBufferSize_M));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrilu02_bufferSize failed with status code: " + stringify(status));
#if CUSPARSE_VER_MAJOR < 12
        status = hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, info->trans_L, m, nnz,
            info->descr_L, csrVal, csrRowPtr, csrColInd, info->info_L, &(info->pBufferSize_L));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseDcsrsv2_bufferSize failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_bufferSize(Util::Intern::cusparse_handle, info->trans_U, m, nnz,
            info->descr_U, csrVal, csrRowPtr, csrColInd, info->info_L, &(info->pBufferSize_U)); //TODO: Error using info_L here?
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseDcsrsv2_bufferSize failed with status code: " + stringify(status));
#else
        const double alpha = 1.;
        status = hipsparseSpSV_bufferSize(Util::Intern::cusparse_handle, info->trans_L, &alpha,
            info->descr_L, NULL /*info->descr_X*/, NULL /*info->descr_Y*/, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, info->info_L, &(info->pBufferSize_L));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSpSV_bufferSize failed with status code: " + stringify(status));

        status = hipsparseSpSV_bufferSize(Util::Intern::cusparse_handle, info->trans_U, &alpha,
            info->descr_U, NULL /*info->descr_X*/, NULL /*info->descr_Y*/, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, info->info_U, &(info->pBufferSize_U));
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSpSV_bufferSize failed with status code: " + stringify(status));
#endif
        info->pBufferSize = max(info->pBufferSize_M, int(max(info->pBufferSize_L, info->pBufferSize_U)));
        info->pBuffer = Util::cuda_malloc(info->pBufferSize_M);

        status = hipsparseDcsrilu02_analysis(Util::Intern::cusparse_handle, m, nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->info_M,
                    info->policy_M, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrilu02_analysis failed with status code: " + stringify(status));
        status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &(info->structural_zero));
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
#if CUSPARSE_VER_MAJOR< 12
        status = hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, info->trans_L, m, nnz, info->descr_L,
                csrVal, csrRowPtr, csrColInd, info->info_L, info->policy_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparse_csrv_analysis failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_analysis(Util::Intern::cusparse_handle, info->trans_U, m, nnz, info->descr_U,
                csrVal, csrRowPtr, csrColInd, info->info_U, info->policy_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparse_csrv_analysis failed with status code: " + stringify(status));
#else
        status = hipsparseSpSV_analysis(Util::Intern::cusparse_handle, info->trans_L, &alpha,
                              info->descr_L, NULL /*info->descr_X*/, NULL /*info->descr_Y*/, HIP_R_64F,
                              HIPSPARSE_SPSV_ALG_DEFAULT, info->info_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSpSV_analysis failed with status code: " + stringify(status));

        status = hipsparseSpSV_analysis(Util::Intern::cusparse_handle, info->trans_U, &alpha,
                              info->descr_U, NULL /*info->descr_X*/, NULL /*info->descr_Y*/, HIP_R_64F,
                              HIPSPARSE_SPSV_ALG_DEFAULT, info->info_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSpSV_analysis failed with status code: " + stringify(status));
#endif

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return (void*)info;
      }

      void cuda_ilu_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        hipsparseStatus_t status = hipsparseDcsrilu02(Util::Intern::cusparse_handle, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->info_M, info->policy_M, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrilu02 failed with status code: " + stringify(status));
        status = hipsparseXcsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &(info->numerical_zero));
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilu_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;
        const double alpha = 1.;
#if CUSPARSE_VER_MAJOR < 12
        hipsparseStatus_t status = hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseDcsrsv2_solve failed with status code: " + stringify(status));

        status = hipsparseDcsrsv2_solve(Util::Intern::cusparse_handle, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsr2_solve failed with status code: " + stringify(status));
#else
        //we have to create vector handlers to use the vector data in the new api (or shift them into our vector handlers... lets see whats necessary)
        hipsparseConstDnVecDescr_t descr_X;
        hipsparseDnVecDescr_t descr_Y, descr_Z;
        hipsparseCreateConstDnVec(&descr_X, info->m, x, HIP_R_64F);
        hipsparseCreateDnVec(&descr_Z, info->m, info->z, HIP_R_64F); //first write into z...
        hipsparseCreateDnVec(&descr_Y, info->m, y, HIP_R_64F);
        //now solve first triang system
        hipsparseStatus_t status = hipsparseSpSV_solve(Util::Intern::cusparse_handle, info->trans_L, &alpha,
                           info->descr_L, descr_X, descr_Z, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, info->info_L);
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
          //delete vecs descr if someone catches the error
          hipsparseDestroyDnVec(descr_Y);
          hipsparseDestroyDnVec(descr_Z);
          hipsparseDestroyDnVec(descr_X);
          throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSpSV_solve failed with status code: " + stringify(status));
        }

        status = hipsparseSpSV_solve(Util::Intern::cusparse_handle, info->trans_U, &alpha,
                           info->descr_U, descr_Z, descr_Y, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, info->info_U);
        if (status != HIPSPARSE_STATUS_SUCCESS)
        {
          hipsparseDestroyDnVec(descr_Y);
          hipsparseDestroyDnVec(descr_Z);
          hipsparseDestroyDnVec(descr_X);
          throw InternalError(__func__, __FILE__, __LINE__, "SECONDcusparseSpSV_solve failed with status code: " + stringify(status));
        }
        //destroy descr
        hipsparseDestroyDnVec(descr_Y);
        hipsparseDestroyDnVec(descr_Z);
        hipsparseDestroyDnVec(descr_X);
#endif

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilu_done_symbolic(void * vinfo)
      {
        CudaIluSolveInfo * info = (CudaIluSolveInfo *) vinfo;

        Util::cuda_free(info->z);
        Util::cuda_free(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
#if CUSPARSE_VER_MAJOR < 12
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
#else
        hipsparseDestroySpMat(info->descr_L);
        hipsparseDestroySpMat(info->descr_U);
#endif
        hipsparseDestroyCsrilu02Info(info->info_M);
#if CUSPARSE_VER_MAJOR < 12
        hipsparseDestroyCsrsv2Info(info->info_L);
        hipsparseDestroyCsrsv2Info(info->info_U);
#else
        hipsparseSpSV_destroyDescr(info->info_L);
        hipsparseSpSV_destroyDescr(info->info_L);
#endif

        delete info;
      }

      // BCSR
      struct CudaIluBSolveInfo
      {
        hipsparseMatDescr_t descr_M;
        hipsparseMatDescr_t descr_L;
        hipsparseMatDescr_t descr_U;
        bsrilu02Info_t info_M;
        bsrsv2Info_t info_L;
        bsrsv2Info_t info_U;
        hipsparseOperation_t trans_L;
        hipsparseOperation_t trans_U;
        hipsparseDirection_t dir;
        hipsparseSolvePolicy_t policy_M;
        hipsparseSolvePolicy_t policy_L;
        hipsparseSolvePolicy_t policy_U;
        void * pBuffer;
        double * z;
        int m;
        int nnz;
        int blocksize;
      };

      void * cuda_ilub_init_symbolic(int m, int nnz, double * csrVal, int * csrRowPtr, int * csrColInd, const int blocksize)
      {
        double * z = (double*)Util::cuda_malloc(m * blocksize * sizeof(double));

        hipsparseMatDescr_t descr_M = 0;
        hipsparseMatDescr_t descr_L = 0;
        hipsparseMatDescr_t descr_U = 0;
        bsrilu02Info_t info_M  = 0;
        bsrsv2Info_t  info_L  = 0;
        bsrsv2Info_t  info_U  = 0;
        int pBufferSize_M;
        int pBufferSize_L;
        int pBufferSize_U;
        int pBufferSize;
        void *pBuffer = 0;
        int structural_zero;
        const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        const hipsparseOperation_t trans_L  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseOperation_t trans_U  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;

        hipsparseStatus_t status;

        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

        hipsparseCreateBsrilu02Info(&info_M);
        hipsparseCreateBsrsv2Info(&info_L);
        hipsparseCreateBsrsv2Info(&info_U);

        hipsparseDbsrilu02_bufferSize(Util::Intern::cusparse_handle, dir, m, nnz,
                descr_M, csrVal, csrRowPtr, csrColInd, blocksize, info_M, &pBufferSize_M);
        hipsparseDbsrsv2_bufferSize(Util::Intern::cusparse_handle, dir, trans_L, m, nnz,
                descr_L, csrVal, csrRowPtr, csrColInd, blocksize, info_L, &pBufferSize_L);
        hipsparseDbsrsv2_bufferSize(Util::Intern::cusparse_handle, dir, trans_U, m, nnz,
                descr_U, csrVal, csrRowPtr, csrColInd, blocksize, info_U, &pBufferSize_U);

        pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

        pBuffer = Util::cuda_malloc(pBufferSize);

        status = hipsparseDbsrilu02_analysis(Util::Intern::cusparse_handle, dir, m, nnz, descr_M,
                csrVal, csrRowPtr, csrColInd, blocksize, info_M,
                    policy_M, pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrilu02 failed with status code: " + stringify(status));
        status = hipsparseXbsrilu02_zeroPivot(Util::Intern::cusparse_handle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }

        status = hipsparseDbsrsv2_analysis(Util::Intern::cusparse_handle, dir, trans_L, m, nnz, descr_L,
                csrVal, csrRowPtr, csrColInd, blocksize, info_L, policy_L, pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrv2_analysis failed with status code: " + stringify(status));

        status = hipsparseDbsrsv2_analysis(Util::Intern::cusparse_handle, dir, trans_U, m, nnz, descr_U,
                csrVal, csrRowPtr, csrColInd, blocksize, info_U, policy_U, pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrv2_analysis failed with status code: " + stringify(status));

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        CudaIluBSolveInfo * info = new CudaIluBSolveInfo;
        info->descr_M = descr_M;
        info->descr_L = descr_L;
        info->descr_U = descr_U;
        info->info_M  = info_M;
        info->info_L  = info_L;
        info->info_U  = info_U;
        info->trans_L = trans_L;
        info->trans_U = trans_U;
        info->dir = dir;
        info->policy_M = policy_M;
        info->policy_L = policy_L;
        info->policy_U = policy_U;
        info->pBuffer = pBuffer;
        info->z = z;
        info->m = m;
        info->nnz = nnz;
        info->blocksize = blocksize;

        return (void*)info;
      }

      void cuda_ilub_init_numeric(double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;

        hipsparseStatus_t status = hipsparseDbsrilu02(Util::Intern::cusparse_handle, info->dir, info->m, info->nnz, info->descr_M,
                csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_M, info->policy_M, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrilu02 failed with status code: " + stringify(status));
        int numerical_zero;
        status = hipsparseXbsrilu02_zeroPivot(Util::Intern::cusparse_handle, info->info_M, &numerical_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        {
          throw InternalError(__func__, __FILE__, __LINE__, "CUSPARSE ZERO PIVOT ERROR!");
        }
      }

      int cuda_ilub_apply(double * y, const double * x, double * csrVal, int * csrRowPtr, int * csrColInd, void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;
        const double alpha = 1.;

        hipsparseStatus_t status = hipsparseDbsrsv2_solve(Util::Intern::cusparse_handle, info->dir, info->trans_L, info->m, info->nnz, &alpha, info->descr_L,
               csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_L,
                  x, info->z, info->policy_L, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrsv2_solve failed with status code: " + stringify(status));

        status = hipsparseDbsrsv2_solve(Util::Intern::cusparse_handle, info->dir, info->trans_U, info->m, info->nnz, &alpha, info->descr_U,
               csrVal, csrRowPtr, csrColInd, info->blocksize, info->info_U,
                  info->z, y, info->policy_U, info->pBuffer);
        if (status != HIPSPARSE_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cusparsebsrsv2_solve failed with status code: " + stringify(status));

        hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
        hipError_t last_error(hipGetLastError());
        if (hipSuccess != last_error)
          throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif

        return 0;
      }

      void cuda_ilub_done_symbolic(void * vinfo)
      {
        CudaIluBSolveInfo * info = (CudaIluBSolveInfo *) vinfo;

        Util::cuda_free(info->z);
        Util::cuda_free(info->pBuffer);
        hipsparseDestroyMatDescr(info->descr_M);
        hipsparseDestroyMatDescr(info->descr_L);
        hipsparseDestroyMatDescr(info->descr_U);
        hipsparseDestroyBsrilu02Info(info->info_M);
        hipsparseDestroyBsrsv2Info(info->info_L);
        hipsparseDestroyBsrsv2Info(info->info_U);

        delete info;
      }
    } // namespace Intern
    /// \endcond
  } // namespace Solver
} // namespace FEAT
