#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2023 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/half.hpp>
#include <kernel/util/string.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/assertion.hpp>

using namespace FEAT;

Index FEAT::Util::cuda_blocksize_misc = 256;
Index FEAT::Util::cuda_blocksize_reduction = 256;
Index FEAT::Util::cuda_blocksize_spmv = 256;
Index FEAT::Util::cuda_blocksize_axpy = 256;

hipsparseHandle_t FEAT::Util::Intern::cusparse_handle;
hipblasHandle_t FEAT::Util::Intern::cublas_handle;
hipblasLtMatmulAlgo_t * FEAT::Util::Intern::cublas_lt_algo_matmat;
bool * FEAT::Util::Intern::cublas_lt_algo_matmat_initialized;
size_t FEAT::Util::Intern::cuda_workspace_size;
void * FEAT::Util::Intern::cuda_workspace;

namespace FEAT
{
  namespace Util
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_set_memory(DT_ * ptr, const DT_ val, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        ptr[idx] = val;
      }

      template <typename DT1_, typename DT2_>
      __global__ void cuda_convert(DT1_ * dest, const DT2_ * src, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        dest[idx] = src[idx];
      }
    }
  }
}

void FEAT::Util::cuda_set_device(const int device)
{
  hipSetDevice(device);
}

void FEAT::Util::cuda_check_last_error()
{
  hipDeviceSynchronize();
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
}

void * FEAT::Util::cuda_get_device_pointer(void * host)
{
  void * device(nullptr);
  if (hipSuccess != hipHostGetDevicePointer((void**)&device, host, 0))
    throw InternalError(__func__, __FILE__, __LINE__, "hipHostGetDevicePointer failed!");
  return device;
}

void * FEAT::Util::cuda_malloc_managed(const Index bytes)
{
  void * memory(nullptr);
  if (bytes == 0)
    return memory;

  auto status = hipMallocManaged((void**)&memory, bytes);
  if (status != hipSuccess)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_malloc_managed allocation error\n" + stringify(hipGetErrorString(status)));
  if (memory == nullptr)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_malloc_managed allocation error (null pointer returned)");
  return memory;
}

void FEAT::Util::cuda_free(void * address)
{
  if (address == nullptr)
    return;

  auto status = hipFree(address);
  if (hipSuccess != status)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_free: hipFree failed!\n" + stringify(hipGetErrorString(status)));
}

void FEAT::Util::cuda_initialize(int rank, int /*ranks_per_node*/, int /*ranks_per_uma*/, int gpus_per_node)
{
  /// \todo enable non cuda ranks and ensure balance of ranks per numa section
  int device = rank % gpus_per_node;
  if (hipSuccess != hipSetDevice(device))
    throw InternalError(__func__, __FILE__, __LINE__, "hipSetDevice failed!");

  int mm_support = 0;
  if (hipSuccess != hipDeviceGetAttribute(&mm_support, hipDeviceAttributeManagedMemory, device))
    throw InternalError(__func__, __FILE__, __LINE__, "cudaGetAttribute failed!");
  XASSERTM(mm_support == 1, "selected cuda device does not support managed memory!");

  if (HIPBLAS_STATUS_SUCCESS != hipblasCreate(&Util::Intern::cublas_handle))
    throw InternalError(__func__, __FILE__, __LINE__, "hipblasCreate failed!");
  if (HIPSPARSE_STATUS_SUCCESS != hipsparseCreate(&Util::Intern::cusparse_handle))
    throw InternalError(__func__, __FILE__, __LINE__, "hipsparseCreate failed!");
  if (HIPBLAS_STATUS_SUCCESS != hipblasSetPointerMode(Util::Intern::cublas_handle, HIPBLAS_POINTER_MODE_HOST))
    throw InternalError(__func__, __FILE__, __LINE__, "hipblasSetPointerMode failed!");
  if (HIPSPARSE_STATUS_SUCCESS != hipsparseSetPointerMode(Util::Intern::cusparse_handle, HIPSPARSE_POINTER_MODE_HOST))
    throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSetPointerMode failed!");

  if (HIPBLAS_STATUS_SUCCESS != hipblasSetMathMode(Util::Intern::cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH))
    throw InternalError(__func__, __FILE__, __LINE__, "hipblasSetMathMode failed!");

  Util::Intern::cublas_lt_algo_matmat = new hipblasLtMatmulAlgo_t[6];
  Util::Intern::cublas_lt_algo_matmat_initialized = new bool[6];
  for (int i(0) ; i < 6 ; ++i)
  {
    Util::Intern::cublas_lt_algo_matmat_initialized[i] = false;
  }

  //Util::Intern::cuda_workspace_size = 1024ul * 1024ul * 1024ul * 2ul;
  Util::Intern::cuda_workspace_size = 0;
  /*auto status = hipMalloc(&(Util::Intern::cuda_workspace), Util::Intern::cuda_workspace_size);
  if (status != hipSuccess)
    throw InternalError(__func__, __FILE__, __LINE__, "hipMalloc failed: " + stringify(hipGetErrorString(status)));*/
}

void FEAT::Util::cuda_finalize()
{
  if (hipSuccess != hipDeviceSynchronize())
    throw InternalError(__func__, __FILE__, __LINE__, "hipDeviceSynchronize failed!");

  if (HIPBLAS_STATUS_SUCCESS != hipblasDestroy(Util::Intern::cublas_handle))
    throw InternalError(__func__, __FILE__, __LINE__, "hipblasDestroy failed!");
  if (HIPSPARSE_STATUS_SUCCESS != hipsparseDestroy(Util::Intern::cusparse_handle))
    throw InternalError(__func__, __FILE__, __LINE__, "hipsparseDestroy failed!");

  delete[] Util::Intern::cublas_lt_algo_matmat;
  delete[] Util::Intern::cublas_lt_algo_matmat_initialized;

  //if (hipSuccess != hipFree(Util::Intern::cuda_workspace))
  //  throw InternalError(__func__, __FILE__, __LINE__, "hipFree failed!");

  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "Pending cuda errors occurred in execution!\n" + stringify(hipGetErrorString(last_error)));

  if (hipSuccess != hipDeviceReset())
    throw InternalError(__func__, __FILE__, __LINE__, "hipDeviceReset failed!");
}

void FEAT::Util::cuda_synchronize()
{
  auto status = hipDeviceSynchronize();
  if (status != hipSuccess)
    throw InternalError(__func__, __FILE__, __LINE__, "hipDeviceSynchronize failed: " + stringify(hipGetErrorString(status)));
}

void FEAT::Util::cuda_reset_device()
{
  auto status = hipDeviceReset();
  if (status != hipSuccess)
    throw InternalError(__func__, __FILE__, __LINE__, "hipDeviceReset failed: " + stringify(hipGetErrorString(status)));
}

void FEAT::Util::cuda_copy(void * dest, const void * src, const Index bytes)
{
  auto status = hipMemcpy(dest, src, bytes, hipMemcpyDefault);
  if (status != hipSuccess)
    throw InternalError(__func__, __FILE__, __LINE__, "hipMemcpy failed: " + stringify(hipGetErrorString(status)));
}

void FEAT::Util::cuda_set_blocksize(Index misc, Index reduction, Index spmv, Index axpy)
{
  FEAT::Util::cuda_blocksize_misc = misc;

  FEAT::Util::cuda_blocksize_reduction = reduction;

  FEAT::Util::cuda_blocksize_spmv = spmv;

  FEAT::Util::cuda_blocksize_axpy = axpy;
}

void FEAT::Util::cuda_reset_algos()
{
  for (int i(0) ; i < 6 ; ++i)
  {
    Util::Intern::cublas_lt_algo_matmat_initialized[i] = false;
  }
}

template <typename DT_>
void FEAT::Util::cuda_set_memory(DT_ * address, const DT_ val, const Index count)
{
  Index blocksize = FEAT::Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((count)/(double)(block.x));
  FEAT::Util::Intern::cuda_set_memory<<<grid, block>>>(address, val, count);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_set_memory failed!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void FEAT::Util::cuda_set_memory(Half * , const Half, const Index);
#endif
template void FEAT::Util::cuda_set_memory(float * , const float, const Index);
template void FEAT::Util::cuda_set_memory(double * , const double, const Index);
template void FEAT::Util::cuda_set_memory(unsigned int * , const unsigned int, const Index);
template void FEAT::Util::cuda_set_memory(unsigned long * , const unsigned long, const Index);
template void FEAT::Util::cuda_set_memory(unsigned long long * , const unsigned long long, const Index);

template <typename DT1_, typename DT2_>
void FEAT::Util::cuda_convert(DT1_ * dest, const DT2_ * src, const Index count)
{
  Index blocksize = FEAT::Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((count)/(double)(block.x));
  FEAT::Util::Intern::cuda_convert<<<grid, block>>>(dest, src, count);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "Util::cuda_convert failed!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void FEAT::Util::cuda_convert<Half, float>(Half *, const float *, const Index);
template void FEAT::Util::cuda_convert<float, Half>(float *, const Half *, const Index);
template void FEAT::Util::cuda_convert<Half, double>(Half *, const double *, const Index);
template void FEAT::Util::cuda_convert<double, Half>(double *, const Half *, const Index);
#endif
template void FEAT::Util::cuda_convert<float, double>(float *, const double *, const Index);
template void FEAT::Util::cuda_convert<double, float>(double *, const float *, const Index);
template void FEAT::Util::cuda_convert<unsigned int, unsigned long>(unsigned int *, const unsigned long *, const Index);
template void FEAT::Util::cuda_convert<unsigned int, unsigned long long>(unsigned int *, const unsigned long long *, const Index);
template void FEAT::Util::cuda_convert<unsigned long, unsigned int>(unsigned long *, const unsigned int *, const Index);
template void FEAT::Util::cuda_convert<unsigned long, unsigned long long>(unsigned long *, const unsigned long long *, const Index);
template void FEAT::Util::cuda_convert<unsigned long long, unsigned int>(unsigned long long *, const unsigned int *, const Index);
template void FEAT::Util::cuda_convert<unsigned long long, unsigned long>(unsigned long long *, const unsigned long *, const Index);
template void FEAT::Util::cuda_convert<unsigned int, double>(unsigned int *, const double *, const Index);
template void FEAT::Util::cuda_convert<unsigned long, double>(unsigned long *, const double *, const Index);
template void FEAT::Util::cuda_convert<unsigned int, float>(unsigned int *, const float *, const Index);
template void FEAT::Util::cuda_convert<unsigned long, float>(unsigned long *, const float *, const Index);

int FEAT::Util::cuda_get_device_count()
{
  int numDevices(-1);
  if (hipSuccess != hipGetDeviceCount(&numDevices))
    throw InternalError(__func__, __FILE__, __LINE__, "hipGetDeviceCount failed!");
  return numDevices;
}

String FEAT::Util::cuda_get_visible_devices()
{
  String result("");
  int numDevices(-1);
  if (hipSuccess != hipGetDeviceCount(&numDevices))
    throw InternalError(__func__, __FILE__, __LINE__, "hipGetDeviceCount failed!");
  result += "Number of visible cuda devices: " + stringify(numDevices) + "\n" ;

  for (int idevice(0); idevice<numDevices; ++idevice)
  {
    // get device properties
    hipDeviceProp_t prop;
    if (hipSuccess != hipGetDeviceProperties (&prop, idevice))
      throw InternalError(__func__, __FILE__, __LINE__, "hipGetDeviceProperties failed!");
    // print out device name and compute capabilities
    result += "Device " + stringify(idevice) + ": " + stringify(prop.name) + "\n";
  }
  return result;
}
