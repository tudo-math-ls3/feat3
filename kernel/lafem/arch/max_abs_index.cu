// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/max_abs_index.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

// includes, CUDA
#include <hipblas.h>

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {

      Index cuda_max_abs_index(const float * x, const Index size)
      {
        int result;
        hipblasStatus_t status;
        status = hipblasIsamax(Util::Intern::cublas_handle, int(size), x, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));
        return (Index)result - 1;
      }

      Index cuda_max_abs_index(const double * x, const Index size)
      {
        int result;
        hipblasStatus_t status;
        status = hipblasIdamax(Util::Intern::cublas_handle, int(size), x, 1, &result);
        if (status != HIPBLAS_STATUS_SUCCESS)
          throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));
        return (Index)result - 1;
      }
    }
  }
}

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
Index MaxAbsIndex::value_cuda(const DT_ * const x, const Index size)
{
  Index result = Intern::cuda_max_abs_index(x, size);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  return result;
}

template Index MaxAbsIndex::value_cuda(const float * const, const Index);
template Index MaxAbsIndex::value_cuda(const double * const, const Index);
