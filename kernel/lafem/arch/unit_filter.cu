#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/unit_filter.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

/// \cond internal
namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_>
      __global__ void cuda_unit_filter_rhs(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;
        v[sv_indices[idx]] = sv_elements[idx];
      }

      template <typename DT_, typename IT_>
      __global__ void cuda_unit_filter_def(DT_ * v, const IT_ * sv_indices, const Index ue)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;
        v[sv_indices[idx]] = DT_(0);
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_, typename IT_>
void UnitFilter::filter_rhs_cuda(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_rhs<<<grid, block>>>(v, sv_elements, sv_indices, ue);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilter::filter_rhs_cuda(float *, const float * const, const std::uint64_t * const, const Index);
template void UnitFilter::filter_rhs_cuda(double *, const double * const, const std::uint64_t * const, const Index);
template void UnitFilter::filter_rhs_cuda(float *, const float * const, const std::uint32_t * const, const Index);
template void UnitFilter::filter_rhs_cuda(double *, const double * const, const std::uint32_t * const, const Index);

template <typename DT_, typename IT_>
void UnitFilter::filter_def_cuda(DT_ * v, const IT_ * const sv_indices, const Index ue)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_def<<<grid, block>>>(v, sv_indices, ue);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilter::filter_def_cuda(float *, const std::uint64_t * const, const Index);
template void UnitFilter::filter_def_cuda(double *, const std::uint64_t * const, const Index);
template void UnitFilter::filter_def_cuda(float *, const std::uint32_t * const, const Index);
template void UnitFilter::filter_def_cuda(double *, const std::uint32_t * const, const Index);

/// \endcond
