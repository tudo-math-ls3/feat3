#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/norm.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/half.hpp>

// includes, CUDA
#include <hipblas.h>

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
DT_ Norm2::value_cuda(const DT_ * const x, const Index size)
{
  hipDataType dt;
  hipDataType et;
  if (typeid(DT_) == typeid(double))
  {
      dt = HIP_R_64F;
      et = HIP_R_64F;
  }
  else if (typeid(DT_) == typeid(float))
  {
      dt = HIP_R_32F;
      et = HIP_R_32F;
  }
#ifdef FEAT_HAVE_HALFMATH
  else if (typeid(DT_) == typeid(Half))
  {
      dt = HIP_R_16F;
      et = HIP_R_32F;
  }
#endif
  else
    throw InternalError(__func__, __FILE__, __LINE__, "unsupported data type!");

  hipblasStatus_t status;
  DT_ result(42.);

  status = hipblasNrm2Ex(Util::Intern::cublas_handle, int(size), x, dt, 1, &result, dt, et);
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  return result;
}

#ifdef FEAT_HAVE_HALFMATH
template Half Norm2::value_cuda(const Half * const, const Index);
#endif
template float Norm2::value_cuda(const float * const, const Index);
template double Norm2::value_cuda(const double * const, const Index);
