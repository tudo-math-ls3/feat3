// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/product_matmat.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>
#include <kernel/util/half.hpp>

#include <hipblas.h>
#include <hipblaslt.h>
#include <hipsparse.h>

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void ProductMatMat::dense_cuda(DT_ * r, const DT_ alpha, const DT_ beta,  const DT_ * const x, const DT_ * const y, const DT_ * const z, const Index rows, const Index columns, const Index inner)
{
  if (r==y || r==x || x==y || z==x || z==y)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda ProductMatMat does not allow r==y or r==x or x==y or z==x or z==y!");

  hipblasStatus_t status;

  // inspired by https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuBLASLt/LtSgemm/sample_cublasLt_LtSgemm.cu

  hipblasLtMatmulDesc_t operationDesc = NULL;
  hipblasLtMatrixLayout_t Rdesc = NULL, Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
  hipblasLtMatmulPreference_t preference = NULL;

  int algo_selector = -1;

  hipDataType dt;
  hipblasComputeType_t ct;
  if (typeid(DT_) == typeid(double))
  {
      dt = HIP_R_64F;
      ct = HIPBLAS_COMPUTE_64F;
      algo_selector = (rows > 1 && columns > 1 && inner > 1) ? 0 : 1;
  }
  else if (typeid(DT_) == typeid(float))
  {
      dt = HIP_R_32F;
#if __CUDA_ARCH__ < 800
      ct = HIPBLAS_COMPUTE_32F;
#else
      ct = HIPBLAS_COMPUTE_32F_FAST_TF32;
#endif
      algo_selector = (rows > 1 && columns > 1 && inner > 1) ? 2 : 3;
  }
#ifdef FEAT_HAVE_HALFMATH
  else if (typeid(DT_) == typeid(Half))
  {
      dt = HIP_R_16F;
      ct = HIPBLAS_COMPUTE_16F;
      algo_selector = (rows > 1 && columns > 1 && inner > 1) ? 4 : 5;
  }
#endif
  else
    throw InternalError(__func__, __FILE__, __LINE__, "unsupported data type!");

  status = hipblasLtMatmulDescCreate(&operationDesc, ct, dt);
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

  hipblasLtOrder_t matrix_order = HIPBLASLT_ORDER_ROW;
  status = hipblasLtMatrixLayoutCreate(&Rdesc, dt, rows, columns, columns);
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));
  status = hipblasLtMatrixLayoutSetAttribute(Rdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrix_order, sizeof(hipblasLtOrder_t));
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));
  hipblasLtMatrixLayoutCreate(&Adesc, dt, rows, inner, inner);
  hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrix_order, sizeof(hipblasLtOrder_t));
  hipblasLtMatrixLayoutCreate(&Bdesc, dt, inner, columns, columns);
  hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrix_order, sizeof(hipblasLtOrder_t));
  if (r!=z)
  {
    hipblasLtMatrixLayoutCreate(&Cdesc, dt, rows, columns, columns);
    hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrix_order, sizeof(hipblasLtOrder_t));
  }
  else // r==z -> in-place multiplication
  {
    Cdesc = Rdesc;
  }

  hipblasLtMatmulHeuristicResult_t algo_check_result;
  if (! FEAT::Util::Intern::cublas_lt_algo_matmat_initialized[algo_selector] ||
      HIPBLAS_STATUS_SUCCESS != cublasLtMatmulAlgoCheck((hipblasLtHandle_t)Util::Intern::cublas_handle, operationDesc, Adesc, Bdesc, Cdesc, Rdesc, &(FEAT::Util::Intern::cublas_lt_algo_matmat[algo_selector]), &algo_check_result))
  {
    int num_algos = 0;
    hipblasLtMatmulHeuristicResult_t heuristic_algos = {};

    status = hipblasLtMatmulPreferenceCreate(&preference);
    if (status != HIPBLAS_STATUS_SUCCESS)
      throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));
    //status = hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &(FEAT::Util::Intern::cuda_workspace_size), sizeof(FEAT::Util::Intern::cuda_workspace_size));
    if (status != HIPBLAS_STATUS_SUCCESS)
      throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

    status = hipblasLtMatmulAlgoGetHeuristic((hipblasLtHandle_t)Util::Intern::cublas_handle, operationDesc, Adesc, Bdesc, Cdesc, Rdesc, preference, 1, &heuristic_algos, &num_algos);
    if (status != HIPBLAS_STATUS_SUCCESS)
      throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

    if (num_algos == 0)
      throw InternalError(__func__, __FILE__, __LINE__, "no algo supports our matrices!");

    FEAT::Util::Intern::cublas_lt_algo_matmat[algo_selector] = heuristic_algos.algo;
    FEAT::Util::Intern::cublas_lt_algo_matmat_initialized[algo_selector] = true;
  }

  hipblasLtMatmulAlgo_t * algo = &(FEAT::Util::Intern::cublas_lt_algo_matmat[algo_selector]);

  //status = hipblasLtMatmul((hipblasLtHandle_t)Util::Intern::cublas_handle, operationDesc, &alpha, x, Adesc, y, Bdesc, &beta, z, Cdesc, r, Rdesc, algo, FEAT::Util::Intern::cuda_workspace, FEAT::Util::Intern::cuda_workspace_size, 0);
  status = hipblasLtMatmul((hipblasLtHandle_t)Util::Intern::cublas_handle, operationDesc, &alpha, x, Adesc, y, Bdesc, &beta, z, Cdesc, r, Rdesc, algo, NULL, 0, 0);
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void ProductMatMat::dense_cuda(Half *, const Half, const Half, const Half * const, const Half * const, const Half * const, const Index, const Index, const Index);
#endif
template void ProductMatMat::dense_cuda(float *, const float, const float, const float * const, const float * const, const float * const, const Index, const Index, const Index);
template void ProductMatMat::dense_cuda(double *, const double, const double, const double * const, const double * const, const double * const, const Index, const Index, const Index);

template <typename DT_, typename IT_>
void ProductMatMat::dsd_cuda(DT_ * r, const DT_ alpha, const DT_ beta, const DT_ * const val, const IT_ * const col_ind, const IT_ * const row_ptr, const Index used_elements,
    const DT_ * y, const Index rows, const Index columns, const Index inner)
{
  if (r==y)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda ProductMatMat does not allow r==y!");

  hipDataType dt;
  hipDataType ct; //compute type
  if (typeid(DT_) == typeid(double))
  {
      dt = HIP_R_64F;
      ct = HIP_R_64F;
  }
  else if (typeid(DT_) == typeid(float))
  {
      dt = HIP_R_32F;
      ct = HIP_R_32F;
  }
#ifdef FEAT_HAVE_HALFMATH
  else if (typeid(DT_) == typeid(Half))
  {
      dt = HIP_R_16F;
      ct = HIP_R_32F; //hipsparseSpMM does not support computation in half, yet
  }
#endif
  else
  {
    throw InternalError(__func__, __FILE__, __LINE__, "unsupported data type!");
  }

  hipsparseIndexType_t it;
  if(sizeof(IT_) == 4u)
    it = HIPSPARSE_INDEX_32I;
  else if(sizeof(IT_) == 8u)
    it = HIPSPARSE_INDEX_64I;
  else
  {
    throw InternalError(__func__, __FILE__, __LINE__, "unsupported index type!");
  }

  hipsparseStatus_t status;

  hipsparseDnMatDescr_t descr_r=0;
  status = hipsparseCreateDnMat(&descr_r, rows, columns, columns, (void*)r, dt, HIPSPARSE_ORDER_ROW);
  if (status != HIPSPARSE_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(hipsparseGetErrorString(status)));

  hipsparseSpMatDescr_t descr_x=0;
  status = hipsparseCreateCsr(&descr_x, rows, inner, used_elements, (void*)row_ptr, (void*)col_ind, (void*)val, it, it, HIPSPARSE_INDEX_BASE_ZERO, dt);
  if (status != HIPSPARSE_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(hipsparseGetErrorString(status)));

  hipsparseDnMatDescr_t descr_y=0;
  status = hipsparseCreateDnMat(&descr_y, inner, columns, columns, (void*)y, dt, HIPSPARSE_ORDER_ROW);
  if (status != HIPSPARSE_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(hipsparseGetErrorString(status)));

  hipsparseOperation_t trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  size_t buffer_size(0);
  status = hipsparseSpMM_bufferSize(Util::Intern::cusparse_handle, trans, trans, &alpha, descr_x, descr_y, &beta, descr_r, ct, HIPSPARSE_SPMM_CSR_ALG2, &buffer_size);
  if (status != HIPSPARSE_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cusparsecsrmvex_buffersize failed with status code: " + stringify(hipsparseGetErrorString(status)));

  void* buffer = Util::cuda_malloc(buffer_size);

  status = hipsparseSpMM(Util::Intern::cusparse_handle, trans, trans, &alpha, descr_x, descr_y, &beta, descr_r, ct, HIPSPARSE_SPMM_CSR_ALG2, buffer);
  if (status != HIPSPARSE_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "hipsparseSpMM failed with status code: " + stringify(hipsparseGetErrorString(status)));

  hipsparseDestroyDnMat(descr_r);
  hipsparseDestroySpMat(descr_x);
  hipsparseDestroyDnMat(descr_y);
  Util::cuda_free(buffer);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void ProductMatMat::dsd_cuda(Half *, const Half, const Half, const Half * const, const std::uint32_t * const, const std::uint32_t * const, const Index, const Half *, const Index, const Index, const Index);
#endif
template void ProductMatMat::dsd_cuda(float *, const float, const float, const float * const, const std::uint32_t * const, const std::uint32_t * const, const Index, const float *, const Index, const Index, const Index);
template void ProductMatMat::dsd_cuda(double *, const double, const double, const double * const, const std::uint32_t * const, const std::uint32_t * const, const Index, const double *, const Index, const Index, const Index);
#ifdef FEAT_HAVE_HALFMATH
template void ProductMatMat::dsd_cuda(Half *, const Half, const Half, const Half * const, const std::uint64_t * const, const std::uint64_t * const, const Index, const Half *, const Index, const Index, const Index);
#endif
template void ProductMatMat::dsd_cuda(float *, const float, const float, const float * const, const std::uint64_t * const, const std::uint64_t * const, const Index, const float *, const Index, const Index, const Index);
template void ProductMatMat::dsd_cuda(double *, const double, const double, const double * const, const std::uint64_t * const, const std::uint64_t * const, const Index, const double *, const Index, const Index, const Index);
