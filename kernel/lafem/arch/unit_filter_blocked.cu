#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/unit_filter_blocked.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

/// \cond internal
namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_rhs(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue, bool ign_nans)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        if(ign_nans)
        {
          for(Index j(0) ; j < block_size; ++j)
          {
            if(!isnan(sv_elements[block_size * idx + j]))
              v[block_size* sv_indices[idx] + j] = sv_elements[block_size * idx + j];
          }
        }
        else
        {
          for(Index j(0) ; j < block_size; ++j)
            v[block_size* sv_indices[idx] + j] = sv_elements[block_size * idx + j];
        }
      }

      template <typename DT_, typename IT_, int BlockSize_>
      __global__ void cuda_unit_filter_blocked_def(DT_ * v, const DT_ * sv_elements, const IT_ * sv_indices, const Index ue, bool ign_nans)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= ue)
          return;

        Index block_size = Index(BlockSize_);
        if(ign_nans)
        {
          for(Index j(0) ; j < block_size; ++j)
          {
            if(!isnan(sv_elements[block_size * idx + j]))
              v[block_size* sv_indices[idx] + j] = DT_(0);
          }
        }
        else
        {
          for(Index j(0) ; j < block_size; ++j)
            v[block_size * sv_indices[idx] + j] = DT_(0);
        }
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <int BlockSize_, typename DT_, typename IT_>
void UnitFilterBlocked::filter_rhs_cuda(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue, bool ign_nans)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_blocked_rhs<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_elements, sv_indices, ue, ign_nans);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked::filter_rhs_cuda<1, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<1, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<1, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<1, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<2, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<2, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<2, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<2, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<3, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<3, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<3, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<3, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<4, float, std::uint64_t>(float *, const float * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<4, double, std::uint64_t>(double *, const double * const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<4, float, std::uint32_t>(float *, const float * const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_rhs_cuda<4, double, std::uint32_t>(double *, const double * const, const std::uint32_t * const, const Index, bool ign_nans);

template <int BlockSize_, typename DT_, typename IT_>
void UnitFilterBlocked::filter_def_cuda(DT_ * v, const DT_ * const sv_elements, const IT_ * const sv_indices, const Index ue, bool ign_nans)
{
  Index blocksize = Util::cuda_blocksize_misc;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((ue)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_unit_filter_blocked_def<DT_, IT_, BlockSize_><<<grid, block>>>(v, sv_elements, sv_indices, ue, ign_nans);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}

template void UnitFilterBlocked::filter_def_cuda<1, float, std::uint64_t>(float *, const float* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<1, double, std::uint64_t>(double *, const double* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<1, float, std::uint32_t>(float *, const float* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<1, double, std::uint32_t>(double *, const double* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<2, float, std::uint64_t>(float *, const float* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<2, double, std::uint64_t>(double *, const double* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<2, float, std::uint32_t>(float *, const float* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<2, double, std::uint32_t>(double *, const double* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<3, float, std::uint64_t>(float *, const float* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<3, double, std::uint64_t>(double *, const double* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<3, float, std::uint32_t>(float *, const float* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<3, double, std::uint32_t>(double *, const double* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<4, float, std::uint64_t>(float *, const float* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<4, double, std::uint64_t>(double *, const double* const, const std::uint64_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<4, float, std::uint32_t>(float *, const float* const, const std::uint32_t * const, const Index, bool ign_nans);
template void UnitFilterBlocked::filter_def_cuda<4, double, std::uint32_t>(double *, const double* const, const std::uint32_t * const, const Index, bool ign_nans);

/// \endcond
