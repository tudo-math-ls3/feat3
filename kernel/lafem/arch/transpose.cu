// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/transpose.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

void Transpose::value_cuda(float * r, const float * const x, Index rows_x, Index columns_x)
{
  hipblasStatus_t status;
  float one(1);
  float zero(0);
  float* temp = nullptr;

  if (r == x)
  {
    temp = (float*)Util::cuda_malloc(rows_x * columns_x * sizeof(float));
    Util::cuda_copy_device_to_device(temp, x, rows_x * columns_x * sizeof(float));
    status = hipblasSgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, int(rows_x), int(columns_x), &one, temp, int(columns_x), &zero, nullptr, int(columns_x), r, int(rows_x));
  }
  else
  {
    status = hipblasSgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, int(rows_x), int(columns_x), &one, x, int(columns_x), &zero, nullptr, int(columns_x), r, int(rows_x));
  }
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));


  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  //free ptr
  Util::cuda_free(temp);
}

void Transpose::value_cuda(double * r, const double * const x, Index rows_x, Index columns_x)
{
  hipblasStatus_t status;
  double one(1);
  double zero(0);
  double *temp = nullptr;

  if (r == x)
  {
    temp = (double*)Util::cuda_malloc(rows_x * columns_x * sizeof(double));
    Util::cuda_copy(temp, x, rows_x * columns_x * sizeof(double));
    status = hipblasDgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, int(rows_x), int(columns_x), &one, temp, int(columns_x), &zero, nullptr, int(columns_x), r, int(rows_x));
  }
  else
  {
    status = hipblasDgeam(Util::Intern::cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, int(rows_x), int(columns_x), &one, x, int(columns_x), &zero, nullptr, int(columns_x), r, int(rows_x));
  }

  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
  Util::cuda_free(temp);
}
