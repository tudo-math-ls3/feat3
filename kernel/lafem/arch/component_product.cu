#include "hip/hip_runtime.h"
// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/component_product.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/memory_pool.hpp>
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/half.hpp>

namespace FEAT
{
  namespace LAFEM
  {
    namespace Intern
    {
      template <typename DT_>
      __global__ void cuda_ComponentProduct(DT_ * r, const DT_ * x, const DT_ * y, const Index count)
      {
        Index idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= count)
          return;
        r[idx] = x[idx] * y[idx];
      }
    }
  }
}


using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void ComponentProduct::value_cuda(DT_ * r, const DT_ * const x, const DT_ * const y, const Index size)
{
  Index blocksize = Util::cuda_blocksize_axpy;
  dim3 grid;
  dim3 block;
  block.x = (unsigned)blocksize;
  grid.x = (unsigned)ceil((size)/(double)(block.x));

  FEAT::LAFEM::Intern::cuda_ComponentProduct<<<grid, block>>>(r, x, y, size);

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void ComponentProduct::value_cuda(Half *, const Half * const, const Half * const, const Index);
#endif
template void ComponentProduct::value_cuda(float *, const float * const, const float * const, const Index);
template void ComponentProduct::value_cuda(double *, const double * const, const double * const, const Index);
