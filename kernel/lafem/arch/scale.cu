// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/scale.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/half.hpp>

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void Scale::value_cuda(DT_ * r, const DT_ * const x, const DT_ s, const Index size)
{
  hipDataType dt;
  hipDataType et;
  if (typeid(DT_) == typeid(double))
  {
      dt = HIP_R_64F;
      et = HIP_R_64F;
  }
  else if (typeid(DT_) == typeid(float))
  {
      dt = HIP_R_32F;
      et = HIP_R_32F;
  }
#ifdef FEAT_HAVE_HALFMATH
  else if (typeid(DT_) == typeid(Half))
  {
      dt = HIP_R_16F;
      et = HIP_R_32F;
  }
#endif
  else
    throw InternalError(__func__, __FILE__, __LINE__, "unsupported data type!");

  if (r != x)
    ///\todo cuse cublasCopyEx when available
    hipMemcpy(r, x, size * sizeof(DT_), hipMemcpyDefault);

  hipblasStatus_t status;

  status = hipblasScalEx(Util::Intern::cublas_handle, int(size), &s, et, r, dt, 1, et);
  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void Scale::value_cuda(Half *, const Half * const, const Half, const Index);
#endif
template void Scale::value_cuda(float *, const float * const, const float, const Index);
template void Scale::value_cuda(double *, const double * const, const double, const Index);
