// FEAT3: Finite Element Analysis Toolbox, Version 3
// Copyright (C) 2010 - 2023 by Stefan Turek & the FEAT group
// FEAT3 is released under the GNU General Public License version 3,
// see the file 'copyright.txt' in the top level directory for details.

// includes, FEAT
#include <kernel/base_header.hpp>
#include <kernel/lafem/arch/axpy.hpp>
#include <kernel/util/exception.hpp>
#include <kernel/util/cuda_util.hpp>
#include <kernel/util/math.hpp>
#include <kernel/util/half.hpp>

using namespace FEAT;
using namespace FEAT::LAFEM;
using namespace FEAT::LAFEM::Arch;

template <typename DT_>
void Axpy::value_cuda(DT_ * r, const DT_ a, const DT_ * const x, const DT_ * const y, const Index size)
{
  hipDataType dt;
  hipDataType et;
  if (typeid(DT_) == typeid(double))
  {
      dt = HIP_R_64F;
      et = HIP_R_64F;
  }
  else if (typeid(DT_) == typeid(float))
  {
      dt = HIP_R_32F;
      et = HIP_R_32F;
  }
#ifdef FEAT_HAVE_HALFMATH
  else if (typeid(DT_) == typeid(Half))
  {
      dt = HIP_R_16F;
      et = HIP_R_32F;
  }
#endif
  else
    throw InternalError(__func__, __FILE__, __LINE__, "unsupported data type!");

  hipblasStatus_t status;
  void* temp_x;

  if (r == x)
  {
    temp_x = Util::cuda_malloc(sizeof(DT_) * size);
    Util::cuda_copy(temp_x, x, size * sizeof(DT_));
  }
  else
  {
    temp_x = (void*)x;
  }

  if (r != y)
  {
    ///\todo cuse cublasCopyEx when available
    Util::cuda_copy(r, y, size * sizeof(DT_));
  }

  status = hipblasAxpyEx(Util::Intern::cublas_handle, int(size), &a, et, temp_x, dt, 1, r, dt, 1, et);
  if (r == x)
    Util::cuda_free(temp_x);

  if (status != HIPBLAS_STATUS_SUCCESS)
    throw InternalError(__func__, __FILE__, __LINE__, "cuda error: " + stringify(cublasGetStatusString(status)));

  hipDeviceSynchronize();
#ifdef FEAT_DEBUG_MODE
  hipError_t last_error(hipGetLastError());
  if (hipSuccess != last_error)
    throw InternalError(__func__, __FILE__, __LINE__, "CUDA error occurred in execution!\n" + stringify(hipGetErrorString(last_error)));
#endif
}
#ifdef FEAT_HAVE_HALFMATH
template void Axpy::value_cuda(Half *, const Half, const Half * const, const Half * const, const Index);
#endif
template void Axpy::value_cuda(float *, const float, const float * const, const float * const, const Index);
template void Axpy::value_cuda(double *, const double, const double * const, const double * const, const Index);
